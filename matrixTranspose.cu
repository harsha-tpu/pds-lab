#include "stdio.h"
#include <hip/hip_runtime.h>


#define N 9
#define WIDTH 3

__global__ void transpose(int *d_m, int *d_t)
{
    int tid = threadIdx.x;
    int row = tid / WIDTH;
    int col = tid % WIDTH;
    
    int transposed_index = col * WIDTH + row;
    d_t[transposed_index] = d_m[tid];
}

int main(void)
{
    int a[N], b[N];
    int *d_m, *d_t;
    
    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    
    hipMalloc((void**) &d_m, N * sizeof(int));
    hipMalloc((void**) &d_t, N * sizeof(int));

    for (int i=0; i<N; i++)
    {
        a[i] = i+1;
    }

    hipMemcpy(d_m, a, N*sizeof(int), hipMemcpyHostToDevice);

    // Record start event
    hipEventRecord(start);
    
    transpose<<<1, N>>>(d_m, d_t);
    
    // Record stop event
    hipEventRecord(stop);
    
    // Wait for kernel to complete
    hipDeviceSynchronize();
    
    // Calculate elapsed time
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(b, d_t, N*sizeof(int), hipMemcpyDeviceToHost);

    printf("Original matrix (3x3):\n");
    for (int i=0; i<N; i++)
    {
        printf("%d\t", a[i]);
        if ((i+1) % WIDTH == 0) printf("\n");
    }

    printf("\nTranspose of the matrix (3x3):\n");
    for (int i=0; i<N; i++)
    {
        printf("%d\t", b[i]);
        if ((i+1) % WIDTH == 0) printf("\n");
    }
    
    // Print execution time
    printf("\nKernel execution time: %.3f milliseconds\n", milliseconds);
    printf("Matrix size: %dx%d (%d elements)\n", WIDTH, WIDTH, N);

    // Destroy events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipFree(d_m);
    hipFree(d_t);

    return 0;
}
