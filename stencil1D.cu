#include <stdio.h> 
#include <hip/hip_runtime.h> 
#define RADIUS 2 
__constant__ int dwgt[2*RADIUS+1];

__global__ void stencil1D(int *in, int *out, const int n) {
  int tid = threadIdx.x;
  int result = 0; 
  for (int i = -RADIUS; i <= RADIUS; i++) {
    if (i >= 0 && i < n) {
      result += in[i]; 
    }
  }
  out[tid] = result; 
}

int main() {
  const int n = 8, width = 2*RADIUS+1; 
  int host[n] = {1, 2, 3, 4, 5, 6, 7, 8};
  int *input, *output; 
  int hwgt[width] = {1, 1, 1, 1, 1};
  hipMalloc((void**)&input, n * sizeof(int));
  hipMalloc((void**)&output, n * sizeof(int));
  hipMemcpy(input, host, n * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(dwgt), hwgt, width * sizeof(int));
  stencil1D<<<1, n>>>(input, output, n);
  hipMemcpy(host, output, n * sizeof(int), hipMemcpyDeviceToHost);
  printf("Input List: ");
  for (int i = 0; i < n; i++)
    printf("%d ", input[i]);
    printf("\n");
  printf("Output List: ");
  for (int i = 0; i < n; i++)
    printf("%d ", input[i]);
    printf("\n");
  return 0; 
}
