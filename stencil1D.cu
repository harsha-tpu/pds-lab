#include "hip/hip_runtime.h"
#include <stdio.h> 
#include <hip/hip_runtime.h> 
#define RADIUS 2 
__constant__ int dwgt[2*RADIUS+1];

__global__ void stencil1D(int *in, int *out, const int n) {
  int tid = threadIdx.x
  if (tid < n) {
    int result = 0; 
    for (int i = -RADIUS; i <= RADIUS; i++) {
      int idx = tid + i;
      if (idx >= 0 && idx < n) {
        result += in[idx] * dwgt[i + RADIUS]; 
      }
    }
    out[tid] = result; 
  }
}

int main() {
  const int n = 8, width = 2*RADIUS+1; 
  int host[n] = {1, 2, 3, 4, 5, 6, 7, 8};
  int *input, *output; 
  int hwgt[width] = {1, 1, 1, 1, 1};
  hipMalloc((void**)&input, n * sizeof(int));
  hipMalloc((void**)&output, n * sizeof(int));
  hipMemcpy(input, host, n * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(dwgt), hwgt, width * sizeof(int));
  stencil1D<<<1, n>>>(input, output, n);
  hipMemcpy(host, output, n * sizeof(int), hipMemcpyDeviceToHost);
  printf("Input List: ");
  for (int i = 0; i < n; i++)
    printf("%d ", input[i]);
    printf("\n");
  printf("Output List: ");
  for (int i = 0; i < n; i++)
    printf("%d ", input[i]);
    printf("\n");

  hipFree(input); 
  hipFree(output);
  return 0; 
}
