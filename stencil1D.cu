#include "hip/hip_runtime.h"
#include <stdio.h> 
#include <hip/hip_runtime.h> 
#define RADIUS 2 
__constant__ int dwgt[2*RADIUS+1];

__global__ void stencil1D(int *in, int *out, const int n) {
  int tid = threadIdx.x;
  if (tid < n) {
    int result = 0; 
    for (int i = -RADIUS; i <= RADIUS; i++) {
      int idx = tid + i;
      if (idx >= 0 && idx < n) {
        result += in[idx] * dwgt[i + RADIUS]; 
      }
    }
    out[tid] = result; 
  }
}

int main() {
  const int n = 8, width = 2*RADIUS+1; 
  int host[n] = {1, 2, 3, 4, 5, 6, 7, 8};
  int *input, *output; 
  int hwgt[width] = {1, 1, 1, 1, 1};
  hipMalloc((void**)&input, n * sizeof(int));
  hipMalloc((void**)&output, n * sizeof(int));
  hipMemcpy(input, host, n * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(dwgt), hwgt, width * sizeof(int));
  stencil1D<<<1, n>>>(input, output, n);
  hipMemcpy(host, output, n * sizeof(int), hipMemcpyDeviceToHost);

  printf("Output List: ");
  for (int i = 0; i < n; i++)
    printf("%d ", host[i]);
    printf("\n");

  hipFree(input); 
  hipFree(output);
  return 0; 
}

//------------------------OUTPUT------------------------

[cse7e23@node1 ~]$ nvcc stencil1D.cu
[cse7e23@node1 ~]$ ./a.out
Output List: 6 10 15 20 25 30 26 21
